#include "hip/hip_runtime.h"
#include "Network.h"

// Block width for CUDA kernels
#define BLOCK_WIDTH 128

/**
 * Computes the backpropagation results of the Softmax loss for each result in a batch.
 * Uses the softmax values obtained from forward propagation to compute the difference.
 *
 * @param label The training batch label values.
 * @param num_labels The number of possible labels.
 * @param batch_size The size of the trained batch.
 * @param diff The resulting gradient.
 */
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    const int label_value = static_cast<int>(label[idx]);

    // For each item in the batch, decrease the result of the label's value by 1
    diff[idx * num_labels + label_value] -= 1.0f;
}

/**
 * Computes ceil(x / y) for integral nonnegative values.
 */
static inline unsigned int RoundUp(unsigned int nominator, unsigned int denominator)
{
    return (nominator + denominator - 1) / denominator;
}

Network::Network(int gpuId, int batchSize,
        ConvolutionalLayer& conv1, MaxPoolingLayer& pool1, ConvolutionalLayer& conv2, MaxPoolingLayer& pool2,
        FullyConnectedLayer& fc1, FullyConnectedLayer& fc2) : ref_fc1(fc1), ref_fc2(fc2), m_gpuid(gpuId) {
    m_batchSize = batchSize;

    // Create CUBLAS and CUDNN handles
    hipSetDevice(gpuId);
    hipblasCreate(&cublasHandle);
    hipdnnCreate(&cudnnHandle);

    // Create tensor descriptors
    hipdnnCreateTensorDescriptor(&dataTensor);
    hipdnnCreateTensorDescriptor(&conv1Tensor);
    hipdnnCreateTensorDescriptor(&conv1BiasTensor);
    hipdnnCreateTensorDescriptor(&pool1Tensor);
    hipdnnCreateTensorDescriptor(&conv2Tensor);
    hipdnnCreateTensorDescriptor(&conv2BiasTensor);
    hipdnnCreateTensorDescriptor(&pool2Tensor);
    hipdnnCreateTensorDescriptor(&fc1Tensor);
    hipdnnCreateTensorDescriptor(&fc2Tensor);

    hipdnnCreateActivationDescriptor(&fc1Activation);

    hipdnnCreateFilterDescriptor(&conv1filterDesc);
    hipdnnCreateFilterDescriptor(&conv2filterDesc);

    hipdnnCreateConvolutionDescriptor(&conv1Desc);
    hipdnnCreateConvolutionDescriptor(&conv2Desc);

    hipdnnCreatePoolingDescriptor(&poolDesc);


    // Set tensor descriptor sizes
    hipdnnSetTensor4dDescriptor(conv1BiasTensor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               1, conv1.getOutputChannels(),
                               1, 1);
    hipdnnSetTensor4dDescriptor(conv2BiasTensor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               1, conv2.getOutputChannels(),
                               1, 1);

    hipdnnSetPooling2dDescriptor(poolDesc,
                                HIPDNN_POOLING_MAX,
                                HIPDNN_PROPAGATE_NAN,
                                pool1.getPoolingSize(), pool1.getPoolingSize(),
                                0, 0,
                                pool1.getStride(), pool1.getStride());
    hipdnnSetTensor4dDescriptor(pool2Tensor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               batchSize, conv2.getOutputChannels(),
                               conv2.getOutputHeight() / pool2.getStride(),
                               conv2.getOutputWidth() / pool2.getStride());

    hipdnnSetTensor4dDescriptor(fc1Tensor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               batchSize, fc1.getOutputs(), 1, 1);

    hipdnnSetTensor4dDescriptor(fc2Tensor,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               batchSize, fc2.getOutputs(), 1, 1);

    hipdnnSetActivationDescriptor(fc1Activation, HIPDNN_ACTIVATION_RELU,
                                 HIPDNN_PROPAGATE_NAN, 0.0);

    // Set convolution tensor sizes and compute workspace size
    size_t workspace = 0;
    workspace = std::max(workspace, SetFwdConvolutionTensors(conv1, dataTensor, conv1Tensor, conv1filterDesc, conv1Desc, conv1algo));
    workspace = std::max(workspace, SetBwdConvolutionTensors(dataTensor, conv1Tensor, conv1filterDesc, conv1Desc, &conv1bwfalgo, nullptr));

    workspace = std::max(workspace, SetFwdConvolutionTensors(conv2, pool1Tensor, conv2Tensor, conv2filterDesc, conv2Desc, conv2algo));
    workspace = std::max(workspace, SetBwdConvolutionTensors(pool1Tensor, conv2Tensor, conv2filterDesc, conv2Desc, &conv2bwfalgo, &conv2bwdalgo));

    // The workspace is allocated later (if necessary)
    m_workspaceSize = workspace;
}

Network::~Network()
{
    hipSetDevice(m_gpuid);

    hipblasDestroy(cublasHandle);
    hipdnnDestroy(cudnnHandle);
    hipdnnDestroyTensorDescriptor(dataTensor);
    hipdnnDestroyTensorDescriptor(conv1Tensor);
    hipdnnDestroyTensorDescriptor(conv1BiasTensor);
    hipdnnDestroyTensorDescriptor(pool1Tensor);
    hipdnnDestroyTensorDescriptor(conv2Tensor);
    hipdnnDestroyTensorDescriptor(conv2BiasTensor);
    hipdnnDestroyTensorDescriptor(pool2Tensor);
    hipdnnDestroyTensorDescriptor(fc1Tensor);
    hipdnnDestroyTensorDescriptor(fc2Tensor);
    hipdnnDestroyActivationDescriptor(fc1Activation);
    hipdnnDestroyFilterDescriptor(conv1filterDesc);
    hipdnnDestroyFilterDescriptor(conv2filterDesc);
    hipdnnDestroyConvolutionDescriptor(conv1Desc);
    hipdnnDestroyConvolutionDescriptor(conv2Desc);
    hipdnnDestroyPoolingDescriptor(poolDesc);
}

void Network::ForwardPropagation(float *data, float *conv1, float *pool1, float *conv2, float *pool2, float *fc1, float *fc1relu,
                        float *fc2, float *result,
                        float *pconv1, float *pconv1bias,
                        float *pconv2, float *pconv2bias,
                        float *pfc1, float *pfc1bias,
                        float *pfc2, float *pfc2bias, void *workspace, float *onevec)
{
    float alpha = 1.0f, beta = 0.0f;
    hipSetDevice(m_gpuid);

    // Conv1 layer
    hipdnnConvolutionForward(cudnnHandle, &alpha, dataTensor,
                            data, conv1filterDesc, pconv1, conv1Desc,
                            conv1algo, workspace, m_workspaceSize, &beta,
                            conv1Tensor, conv1);
    hipdnnAddTensor(cudnnHandle, &alpha, conv1BiasTensor,
                   pconv1bias, &alpha, conv1Tensor, conv1);

    // Pool1 layer
    hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, conv1Tensor,
                        conv1, &beta, pool1Tensor, pool1);

    // Conv2 layer
    hipdnnConvolutionForward(cudnnHandle, &alpha, pool1Tensor,
                            pool1, conv2filterDesc, pconv2, conv2Desc,
                            conv2algo, workspace, m_workspaceSize, &beta,
                            conv2Tensor, conv2);
    hipdnnAddTensor(cudnnHandle, &alpha, conv2BiasTensor,
                   pconv2bias, &alpha, conv2Tensor, conv2);

    // Pool2 layer
    hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, conv2Tensor,
                        conv2, &beta, pool2Tensor, pool2);

    // FC1 layer
    // Forward propagate neurons using weights (fc1 = pfc1'*pool2)
    hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                ref_fc1.getOutputs(), m_batchSize, ref_fc1.getInputs(),
                &alpha,
                pfc1, ref_fc1.getInputs(),
                pool2, ref_fc1.getInputs(),
                &beta,
                fc1, ref_fc1.getOutputs());
    // Add bias using GEMM's "beta" (fc1 += pfc1bias*1_vec')
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                ref_fc1.getOutputs(), m_batchSize, 1,
                &alpha,
                pfc1bias, ref_fc1.getOutputs(),
                onevec, 1,
                &alpha,
                fc1, ref_fc1.getOutputs());

    // ReLU activation
    hipdnnActivationForward(cudnnHandle, fc1Activation, &alpha,
                           fc1Tensor, fc1, &beta, fc1Tensor, fc1relu);

    // FC2 layer
    // Forward propagate neurons using weights (fc2 = pfc2'*fc1relu)
    hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                ref_fc2.getOutputs(), m_batchSize, ref_fc2.getInputs(),
                &alpha,
                pfc2, ref_fc2.getInputs(),
                fc1relu, ref_fc2.getInputs(),
                &beta,
                fc2, ref_fc2.getOutputs());
    // Add bias using GEMM's "beta" (fc2 += pfc2bias*1_vec')
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                ref_fc2.getOutputs(), m_batchSize, 1,
                &alpha,
                pfc2bias, ref_fc2.getOutputs(),
                onevec, 1,
                &alpha,
                fc2, ref_fc2.getOutputs());

    // Softmax loss
    hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha, fc2Tensor, fc2, &beta, fc2Tensor, result);
}

void Network::Backpropagation(ConvolutionalLayer& layer_conv1, MaxPoolingLayer& layer_pool1, ConvolutionalLayer& layer_conv2, MaxPoolingLayer& layer_pool2,
                     float *data, float *labels, float *conv1, float *pool1, float *conv2, float *pool2, float *fc1, float *fc1relu,
                     float *fc2, float *fc2smax, float *dloss_data,
                     float *pconv1, float *pconv1bias,
                     float *pconv2, float *pconv2bias,
                     float *pfc1, float *pfc1bias,
                     float *pfc2, float *pfc2bias,
                     float *gconv1, float *gconv1bias, float *dpool1,
                     float *gconv2, float *gconv2bias, float *dconv2, float *dpool2,
                     float *gfc1, float *gfc1bias, float *dfc1, float *dfc1relu,
                     float *gfc2, float *gfc2bias, float *dfc2,
                     void *workspace, float *onevec)
{
    float alpha = 1.0f, beta = 0.0f;

    float scalVal = 1.0f / static_cast<float>(m_batchSize);

    hipSetDevice(m_gpuid);

    // Initialization (using the training error function)
    hipMemcpyAsync(dloss_data, fc2smax, sizeof(float) * m_batchSize * ref_fc2.getOutputs(), hipMemcpyDeviceToDevice);

    // Softmax layer
    SoftmaxLossBackprop<<<RoundUp(m_batchSize, BLOCK_WIDTH), BLOCK_WIDTH>>>(labels, ref_fc2.getOutputs(), m_batchSize, dloss_data);

    // Accounting for batch size in SGD
    hipblasSscal(cublasHandle, ref_fc2.getOutputs() * m_batchSize, &scalVal, dloss_data, 1);

    // FC2 layer
    // Compute derivative with respect to weights: gfc2 = (fc1relu * dfc2smax')
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_fc2.getInputs(), ref_fc2.getOutputs(), m_batchSize,
                &alpha, fc1relu, ref_fc2.getInputs(), dloss_data, ref_fc2.getOutputs(), &beta, gfc2, ref_fc2.getInputs());
    // Compute derivative with respect to bias: gfc2bias = dfc2smax * 1_vec
    hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_fc2.getOutputs(), m_batchSize,
                &alpha, dloss_data, ref_fc2.getOutputs(), onevec, 1, &beta, gfc2bias, 1);
    // Compute derivative with respect to data (for previous layer): pfc2*dfc2smax (500x10*10xN)
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_fc2.getInputs(), m_batchSize, ref_fc2.getOutputs(),
                &alpha, pfc2, ref_fc2.getInputs(), dloss_data, ref_fc2.getOutputs(), &beta, dfc2, ref_fc2.getInputs());

    // ReLU activation
    hipdnnActivationBackward(cudnnHandle, fc1Activation, &alpha,
                            fc1Tensor, fc1relu, fc1Tensor, dfc2,
                            fc1Tensor, fc1, &beta, fc1Tensor, dfc1relu);

    // FC1 layer
    // Compute derivative with respect to weights: gfc1 = (pool2 * dfc1relu')
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, ref_fc1.getInputs(), ref_fc1.getOutputs(), m_batchSize,
                &alpha, pool2, ref_fc1.getInputs(), dfc1relu, ref_fc1.getOutputs(), &beta, gfc1, ref_fc1.getInputs());
    // Compute derivative with respect to bias: gfc1bias = dfc1relu * 1_vec
    hipblasSgemv(cublasHandle, HIPBLAS_OP_N, ref_fc1.getOutputs(), m_batchSize,
                &alpha, dfc1relu, ref_fc1.getOutputs(), onevec, 1, &beta, gfc1bias, 1);
    // Compute derivative with respect to data (for previous layer): pfc1*dfc1relu (800x500*500xN)
    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, ref_fc1.getInputs(), m_batchSize, ref_fc1.getOutputs(),
                &alpha, pfc1, ref_fc1.getInputs(), dfc1relu, ref_fc1.getOutputs(), &beta, dfc1, ref_fc1.getInputs());

    // Pool2 layer
    hipdnnPoolingBackward(cudnnHandle, poolDesc, &alpha,
                         pool2Tensor, pool2, pool2Tensor, dfc1,
                         conv2Tensor, conv2, &beta, conv2Tensor, dpool2);

    // Conv2 layer
    hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, conv2Tensor,
                                 dpool2, &beta, conv2BiasTensor, gconv2bias);


    hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, pool1Tensor,
                                   pool1, conv2Tensor, dpool2, conv2Desc,
                                   conv2bwfalgo, workspace, m_workspaceSize,
                                   &beta, conv2filterDesc, gconv2);

    hipdnnConvolutionBackwardData(cudnnHandle, &alpha, conv2filterDesc,
                                 pconv2, conv2Tensor, dpool2, conv2Desc,
                                 conv2bwdalgo, workspace, m_workspaceSize,
                                 &beta, pool1Tensor, dconv2);

    // Pool1 layer
    hipdnnPoolingBackward(cudnnHandle, poolDesc, &alpha,
                         pool1Tensor, pool1, pool1Tensor, dconv2,
                         conv1Tensor, conv1, &beta, conv1Tensor, dpool1);

    // Conv1 layer
    hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, conv1Tensor,
                                 dpool1, &beta, conv1BiasTensor, gconv1bias);

    hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, dataTensor,
                                   data, conv1Tensor, dpool1, conv1Desc,
                                   conv1bwfalgo, workspace, m_workspaceSize,
                                   &beta, conv1filterDesc, gconv1);

    // No need for convBackwardData because there are no more layers below
}

void Network::UpdateWeights(float learning_rate,
                   ConvolutionalLayer& conv1, ConvolutionalLayer& conv2,
                   float *pconv1, float *pconv1bias,
                   float *pconv2, float *pconv2bias,
                   float *pfc1, float *pfc1bias,
                   float *pfc2, float *pfc2bias,
                   float *gconv1, float *gconv1bias,
                   float *gconv2, float *gconv2bias,
                   float *gfc1, float *gfc1bias,
                   float *gfc2, float *gfc2bias)
{
    float alpha = -learning_rate;

    hipSetDevice(m_gpuid);

    // Conv1
    hipblasSaxpy(cublasHandle, static_cast<int>(conv1.getPconv().size()),
                &alpha, gconv1, 1, pconv1, 1);
    hipblasSaxpy(cublasHandle, static_cast<int>(conv1.getPbias().size()),
                &alpha, gconv1bias, 1, pconv1bias, 1);

    // Conv2
    hipblasSaxpy(cublasHandle, static_cast<int>(conv2.getPconv().size()),
                &alpha, gconv2, 1, pconv2, 1);
    hipblasSaxpy(cublasHandle, static_cast<int>(conv2.getPbias().size()),
                &alpha, gconv2bias, 1, pconv2bias, 1);

    // Fully connected 1
    hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.getPneurons().size()),
                &alpha, gfc1, 1, pfc1, 1);
    hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc1.getPbias().size()),
                &alpha, gfc1bias, 1, pfc1bias, 1);

    // Fully connected 2
    hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc2.getPneurons().size()),
                &alpha, gfc2, 1, pfc2, 1);
    hipblasSaxpy(cublasHandle, static_cast<int>(ref_fc2.getPbias().size()),
                &alpha, gfc2bias, 1, pfc2bias, 1);
}

size_t Network::SetFwdConvolutionTensors(ConvolutionalLayer& conv, hipdnnTensorDescriptor_t& srcTensorDesc, hipdnnTensorDescriptor_t& dstTensorDesc,
                                hipdnnFilterDescriptor_t& filterDesc, hipdnnConvolutionDescriptor_t& convDesc,
                                hipdnnConvolutionFwdAlgo_t& algo)
{
    size_t sizeInBytes = 0;

    int n = m_batchSize;
    int c = conv.getInputChannels();
    int h = conv.getInputHeight();
    int w = conv.getInputWidth();

    hipdnnSetTensor4dDescriptor(srcTensorDesc,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               n, c,
                               h, w);

    hipdnnSetFilter4dDescriptor(filterDesc,
                               HIPDNN_DATA_FLOAT,
                               HIPDNN_TENSOR_NCHW,
                               conv.getOutputChannels(),
                               conv.getInputChannels(),
                               conv.getFilterSize(),
                               conv.getFilterSize());

#if CUDNN_MAJOR > 5
    hipdnnSetConvolution2dDescriptor(convDesc,
                                    0, 0,
                                    1, 1,
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
#else
    hipdnnSetConvolution2dDescriptor(convDesc,
                                                   0, 0,
                                                   1, 1,
                                                   1, 1,
                                                   HIPDNN_CROSS_CORRELATION);
#endif

    // Find dimension of convolution output
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          srcTensorDesc,
                                          filterDesc,
                                          &n, &c, &h, &w);

    hipdnnSetTensor4dDescriptor(dstTensorDesc,
                               HIPDNN_TENSOR_NCHW,
                               HIPDNN_DATA_FLOAT,
                               n, c,
                               h, w);
    hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
                                        srcTensorDesc,
                                        filterDesc,
                                        convDesc,
                                        dstTensorDesc,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        0,
                                        &algo);

    hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
                                            srcTensorDesc,
                                            filterDesc,
                                            convDesc,
                                            dstTensorDesc,
                                            algo,
                                            &sizeInBytes);

    return sizeInBytes;
}

size_t Network::SetBwdConvolutionTensors(hipdnnTensorDescriptor_t& srcTensorDesc, hipdnnTensorDescriptor_t& dstTensorDesc,
                                         hipdnnFilterDescriptor_t& filterDesc, hipdnnConvolutionDescriptor_t& convDesc,
                                         hipdnnConvolutionBwdFilterAlgo_t *falgo, hipdnnConvolutionBwdDataAlgo_t *dalgo)
{
    size_t sizeInBytes = 0, tmpsize = 0;

    // If backprop filter algorithm was requested
    if (falgo)
    {
        hipdnnGetConvolutionBackwardFilterAlgorithm(
                cudnnHandle, srcTensorDesc, dstTensorDesc, convDesc, filterDesc,
                HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, falgo);

        hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                cudnnHandle, srcTensorDesc, dstTensorDesc, convDesc, filterDesc,
                *falgo, &tmpsize);

        sizeInBytes = std::max(sizeInBytes, tmpsize);
    }

    // If backprop data algorithm was requested
    if (dalgo)
    {
        hipdnnGetConvolutionBackwardDataAlgorithm(
                cudnnHandle, filterDesc, dstTensorDesc, convDesc, srcTensorDesc,
                HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, dalgo);

        hipdnnGetConvolutionBackwardDataWorkspaceSize(
                cudnnHandle, filterDesc, dstTensorDesc, convDesc, srcTensorDesc,
                *dalgo, &tmpsize);

        sizeInBytes = std::max(sizeInBytes, tmpsize);
    }

    return sizeInBytes;
}

int Network::getBatchSize() const {
    return m_batchSize;
}

size_t Network::getWorkspaceSize() const {
    return m_workspaceSize;
}
